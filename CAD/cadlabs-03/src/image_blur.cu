#include <wb.h>


#define BLUR_SIZE 5

//@@ INSERT CODE HERE

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;


 /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  inputImage = wbImport(inputImageFile);

  // The input image is in grayscale, so the number of channels
  // is 1
  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);

  // Since the image is monochromatic, it only contains only one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  
  //@@ INSERT CODE HERE


  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  
  //@@ INSERT CODE HERE


  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");

//@@ INSERT CODE HERE


  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  
  //@@ INSERT CODE HERE

  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");


  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
