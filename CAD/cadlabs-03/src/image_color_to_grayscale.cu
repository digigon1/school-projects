#include <wb.h>


//@@ INSERT CODE HERE

int main(int argc, char *argv[]) {

  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;

  /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  
  // For this lab the value is always 3
  imageChannels = wbImage_getChannels(inputImage);

  // Since the image is monochromatic, it only contains one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");

 //TODO @@ INSERT CODE HERE



  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");

   //TODO @@ INSERT CODE HERE

  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  //TODO @@ INSERT CODE HERE

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  
 //TODO @@ INSERT CODE HERE

  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");


  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
